#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "kernel.hpp"
#include <math.h>

__global__ void calc(ComplexNumber *input, double *output, int *p_max_iterations, int *p_image_points)
{
    int max_iterations = *p_max_iterations;
    int image_points = *p_image_points;
    int limit = 2; // refractor

    int id = id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < image_points)
    {
        ComplexNumber c = input[id];
        ComplexNumber z = {0.0, 0.0};
        for (int i = 0; i < max_iterations; i++)
        {
            if ((z.a * z.a + z.b * z.b) > limit * limit)
            {
                // four more iterations to smooth it out
                for (int j = 0; j < 4; j++)
                {
                    double newA = z.a * z.a - z.b * z.b + c.a;
                    double newB = 2 * z.a * z.b + c.b;
                    z.a = newA;
                    z.b = newB;
                }
                double modulus = std::sqrt(z.a * z.a + z.b * z.b);
                double smooth_iterations = 5 + i - std::log(std::log(modulus)) / std::log(2);

                output[id] = smooth_iterations;
                return;
            }
            else
            {
                double newA = z.a * z.a - z.b * z.b + c.a;
                double newB = 2 * z.a * z.b + c.b;
                z.a = newA;
                z.b = newB;
            }
        }

        output[id] = 0;
    }
}

void startThreads(ComplexNumber *input, double *output, int *max_iterations, int *image_points)
{
    // Create device pointers
    ComplexNumber *d_input;
    double *d_output;
    int *d_max_iterations, *d_image_points;

    // Calculate grid
    int threads = 256;
    int blocks = std::ceil(*image_points / threads);

    std::cout << "Using " << blocks << " x " << threads << "  for " << *image_points << std::endl;

    // Allocate memory to the device pointers on the GPU
    hipMalloc(&d_input, sizeof(ComplexNumber) * *image_points);
    hipMalloc(&d_output, sizeof(double) * *image_points);
    hipMalloc(&d_max_iterations, sizeof(int));
    hipMalloc(&d_image_points, sizeof(int));

    // Copy data into the allocated memory
    hipMemcpy(d_input, input, sizeof(ComplexNumber) * *image_points, hipMemcpyHostToDevice);
    hipMemcpy(d_max_iterations, max_iterations, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_image_points, image_points, sizeof(int), hipMemcpyHostToDevice);

    // Execute kernel
    calc<<<blocks, threads>>>(d_input, d_output, d_max_iterations, d_image_points);

    // Sync ??????????????????????????????
    hipDeviceSynchronize();

    // Copy the output back to the host
    hipMemcpy(output, d_output, sizeof(double) * *image_points, hipMemcpyDeviceToHost);

    // Free memory
    hipFree(&d_input);
    hipFree(&d_output);
    hipFree(&d_max_iterations);
    hipFree(&d_image_points);

    // Reset the GPU
    hipDeviceReset();
}