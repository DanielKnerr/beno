#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "renderer.hpp"
#include <math.h>
#include <stdio.h>
#include "definitions.hpp"

__global__ void normalized_iterations_count(ComplexNumberGPU *input, double *output, unsigned int *limit, unsigned int *p_max_iterations, unsigned int *p_image_points)
{
    int max_iterations = *p_max_iterations;
    int image_points = *p_image_points;
    int id = id = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (id < image_points)
    {
        ComplexNumberGPU c = input[id];
        ComplexNumberGPU z = {0.0, 0.0};
        for (int i = 0; i < max_iterations; i++)
        {
            if ((z.a * z.a + z.b * z.b) > *limit * *limit)
            {
                // four more iterations to smooth it out
                for (int j = 0; j < 4; j++)
                {
                    double newA = z.a * z.a - z.b * z.b + c.a;
                    double newB = 2 * z.a * z.b + c.b;
                    z.a = newA;
                    z.b = newB;
                }
                double modulus = std::sqrt(z.a * z.a + z.b * z.b);
                double smooth_iterations = 5 + i - std::log(std::log(modulus)) / std::log(2);

                output[id] = smooth_iterations;
                return;
            }
            else
            {
                double newA = z.a * z.a - z.b * z.b + c.a;
                double newB = 2 * z.a * z.b + c.b;
                z.a = newA;
                z.b = newB;
            }
        }

        output[id] = 0;
    }
}

void startThreadsGPU(ComplexNumberMPF *start_point, ComplexNumberMPF *step_size, double *output, Config *config)
{
    mpf_t r, i;
    mpf_init(r);
    mpf_init(i);
    // Convert the MPF starting point to image points
    int image_points = config->image_height * config->image_width;
    ComplexNumberGPU *input = new ComplexNumberGPU[image_points];
    
    
    for (unsigned int y = 0; y < config->image_height; y++) {
        for (unsigned int x = 0; x < config->image_width; x++) {
            int point_idx = y * config->image_width + x;
            int idx_i = point_idx / config->image_width;
            int idx_r = point_idx - idx_i * config->image_width;

            mpf_set_ui(i, 0);
            mpf_set_ui(r, 0);

            mpf_mul_ui(r, (*step_size).r, idx_r);
            mpf_add(r, r, (*start_point).r);

            mpf_mul_ui(i, (*step_size).i, idx_i);
            mpf_add(i, i, (*start_point).i);

            double a = mpf_get_d(i);
            double b = mpf_get_d(r);
            
            input[y * config->image_width + x] = ComplexNumberGPU{b, a};
        }
    }

    // Create device pointers
    ComplexNumberGPU *d_input;
    double *d_output;
    unsigned int *d_max_iterations, *d_image_points, *d_limit;

    // Calculate grid
    int threads = 256;
    int blocks = std::ceil(image_points / threads);

    // Allocate memory to the device pointers on the GPU
    hipMalloc(&d_input, sizeof(ComplexNumberGPU) * image_points);
    hipMalloc(&d_output, sizeof(double) * image_points);
    hipMalloc(&d_max_iterations, sizeof(unsigned int));
    hipMalloc(&d_image_points, sizeof(unsigned int));
    hipMalloc(&d_limit, sizeof(unsigned int));

    // Copy data into the allocated memory
    hipMemcpy(d_input, input, sizeof(ComplexNumberGPU) * image_points, hipMemcpyHostToDevice);
    hipMemcpy(d_max_iterations, &config->max_iterations, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_image_points, &image_points, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_limit, &config->limit, sizeof(unsigned int), hipMemcpyHostToDevice);

    // Execute kernel
    normalized_iterations_count<<<blocks, threads>>>(d_input, d_output, d_limit, d_max_iterations, d_image_points);

    // Sync ??????????????????????????????
    hipDeviceSynchronize();

    // Copy the output back to the host
    hipMemcpy(output, d_output, sizeof(double) * image_points, hipMemcpyDeviceToHost);

    // Free memory
    hipFree(&d_input);
    hipFree(&d_output);
    hipFree(&d_max_iterations);
    hipFree(&d_image_points);

    // Reset the GPU
    hipDeviceReset();
    
    printf("GPPPPUUUU\n");
}