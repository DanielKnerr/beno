#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "kernel.hpp"
#include <math.h>

__global__ void normalized_iterations_count(ComplexNumber *input, double *output, unsigned int *limit, unsigned int *p_max_iterations, unsigned int *p_image_points)
{
    int max_iterations = *p_max_iterations;
    int image_points = *p_image_points;

    int id = id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < image_points)
    {
        ComplexNumber c = input[id];
        ComplexNumber z = {0.0, 0.0};
        for (int i = 0; i < max_iterations; i++)
        {
            if ((z.a * z.a + z.b * z.b) > *limit * *limit)
            {
                // four more iterations to smooth it out
                for (int j = 0; j < 4; j++)
                {
                    double newA = z.a * z.a - z.b * z.b + c.a;
                    double newB = 2 * z.a * z.b + c.b;
                    z.a = newA;
                    z.b = newB;
                }
                double modulus = std::sqrt(z.a * z.a + z.b * z.b);
                double smooth_iterations = 5 + i - std::log(std::log(modulus)) / std::log(2);

                output[id] = smooth_iterations;
                return;
            }
            else
            {
                double newA = z.a * z.a - z.b * z.b + c.a;
                double newB = 2 * z.a * z.b + c.b;
                z.a = newA;
                z.b = newB;
            }
        }

        output[id] = 0;
    }
}

void startThreadsGPU(ComplexNumber *input, double *output, unsigned int *limit, unsigned int *max_iterations, unsigned int *image_points)
{
    // Create device pointers
    ComplexNumber *d_input;
    double *d_output;
    unsigned int *d_max_iterations, *d_image_points, *d_limit;

    // Calculate grid
    int threads = 256;
    int blocks = std::ceil(*image_points / threads);

    // Allocate memory to the device pointers on the GPU
    hipMalloc(&d_input, sizeof(ComplexNumber) * *image_points);
    hipMalloc(&d_output, sizeof(double) * *image_points);
    hipMalloc(&d_max_iterations, sizeof(unsigned int));
    hipMalloc(&d_image_points, sizeof(unsigned int));
    hipMalloc(&d_limit, sizeof(unsigned int));

    // Copy data into the allocated memory
    hipMemcpy(d_input, input, sizeof(ComplexNumber) * *image_points, hipMemcpyHostToDevice);
    hipMemcpy(d_max_iterations, max_iterations, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_image_points, image_points, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_limit, limit, sizeof(unsigned int), hipMemcpyHostToDevice);

    // Execute kernel
    normalized_iterations_count<<<blocks, threads>>>(d_input, d_output, d_limit, d_max_iterations, d_image_points);

    // Sync ??????????????????????????????
    hipDeviceSynchronize();

    // Copy the output back to the host
    hipMemcpy(output, d_output, sizeof(double) * *image_points, hipMemcpyDeviceToHost);

    // Free memory
    hipFree(&d_input);
    hipFree(&d_output);
    hipFree(&d_max_iterations);
    hipFree(&d_image_points);

    // Reset the GPU
    hipDeviceReset();
}